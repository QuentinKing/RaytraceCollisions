#include "hip/hip_runtime.h"
#include "RayStructs.h"
#include "BufferStructs.h"

// Ray data
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float, closestHitDist, rtIntersectionDistance, );
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(int, max_depth, , );

// Camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float, fov, , );

// Output buffers
rtBuffer<uchar4, 2> output_buffer;
rtBuffer<IntersectionResponse, 2> collisionResponse;

// Rigidbody variables
rtDeclareVariable(int, physicsRayStep, , );
rtDeclareVariable(int, physicsBufferWidth, , );
rtDeclareVariable(int, physicsBufferHeight, , );

// Volumetric variables
rtDeclareVariable(IntersectionData, intersectionData, attribute intersectionData, );

// Shading values
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );

// Light values
rtDeclareVariable(float3, ambientLightColor, , );
rtBuffer<Light> lights; 

// Material values
rtDeclareVariable(float3, ambientColorIntensity, , );
rtDeclareVariable(float3, diffuseColorIntensity, , );
rtDeclareVariable(float3, specularColorIntensity, , );
rtDeclareVariable(float3, fresnel, , );
rtDeclareVariable(float3, reflectivity, , );
rtDeclareVariable(float,   specularPower, , );

// Scene values
rtTextureSampler<float4, 2> envmap;

void ClearResponseBuffer()
{
	IntersectionResponse data;
	data.volume = 0.0f;
	data.entryId = 0;
	data.entryNormal = make_float3(0.0f,0.0f,0.0f);
	data.exitId = 0;
	data.exitNormal = make_float3(0.0f,0.0f,0.0f);
	data.entryPoint = make_float3(0.0f,0.0f,0.0f);
	data.exitPoint = make_float3(0.0f,0.0f,0.0f);
	collisionResponse[make_uint2(launch_index.x / physicsRayStep, launch_index.y / physicsRayStep)] = data;
}

// Given an ordered list of ray intersections, finds all intervals of intersections and
// computes their volumes.
void CheckIntersectionOverlap(PerRayData_radiance prd, float3 ray_origin, float3 ray_direction, float3& result)
{
	// Right now, just store the largest intersection volume into the buffer
	IntersectionResponse largestResponse;
	largestResponse.volume = 0.0f;
	largestResponse.entryId = 0;
	largestResponse.entryNormal = make_float3(0.0, 0.0, 0.0);
	largestResponse.exitId = 0;
	largestResponse.exitNormal = make_float3(0.0, 0.0, 0.0);
	largestResponse.entryPoint = make_float3(0.0, 0.0, 0.0);
	largestResponse.exitPoint = make_float3(0.0, 0.0, 0.0);
	largestResponse.collisionId = 0;
	IntersectionData objectsInside[5]; // Assume we will never have more than 5 bodies intersecting at any given point
	int insideIndex = 0;

	float2 screen = make_float2(output_buffer.size());
	float fovDelta = 1.0 / screen.x;
	float theta = fov * fovDelta;
	float phi = 90.0 - theta;

	// Debugging
	float largestT = 0.0;

	for (int i = 0; i < prd.numIntersections; i++)
	{
		IntersectionData objEnter; // Will be set if we need it

		// Check to see if we are entering this object
		bool entering = true;
		for (int j = 0; j < insideIndex; j++)
		{
			if (objectsInside[j].rigidBodyId == prd.intersections[i].rigidBodyId)
			{
				objEnter = objectsInside[j];
				entering = false;
			}
		}

		// If entering, add it to our tracking array
		if (entering)
		{
			// Check if this intersection has an exit point (ie, is valid)
			bool isValid = false;
			for (int j = i + 1; j < prd.numIntersections; j++)
			{
				if (prd.intersections[j].rigidBodyId == prd.intersections[i].rigidBodyId)
				{
					isValid = true;
					break;
				}
			}

			if (isValid)
			{
				objectsInside[insideIndex] = prd.intersections[i];
				insideIndex++;
			}
		}
		else
		{
			// Otherwise, we are exiting this object, need to check for intersection volumes
			// with any other objects we are currently inside
			for (int j = 0; j < insideIndex; j++)
			{
				if (objectsInside[j].rigidBodyId != prd.intersections[i].rigidBodyId)
				{
					// Compute volume
					IntersectionData entryPoint = objectsInside[j].t < objEnter.t ? objEnter : objectsInside[j];
					IntersectionData exitPoint = prd.intersections[i];
					
					float a = sin(theta) * entryPoint.t/ sin(phi);
					float b = sin(theta) * exitPoint.t/ sin(phi);
					float h = exitPoint.t - entryPoint.t;
					float volume = 0.33 * (a*a + a * b + b * b) * h;

					if (volume > largestResponse.volume)
					{
						largestResponse.volume = volume;
						largestResponse.entryId = entryPoint.rigidBodyId;
						largestResponse.entryNormal = entryPoint.normal;
						largestResponse.exitId = exitPoint.rigidBodyId;
						largestResponse.exitNormal = exitPoint.normal;
						largestResponse.entryPoint = ray_origin + entryPoint.t * ray_direction;
						largestResponse.exitPoint = ray_origin + exitPoint.t * ray_direction;
						largestResponse.collisionId = objectsInside[j].rigidBodyId;
					}
				}
			}

			// Remove this object from our tracking array
			bool shift = false;
			for (int j = 0; j < insideIndex; j++)
			{
				if (objectsInside[j].rigidBodyId == prd.intersections[i].rigidBodyId)
				{
					shift = true;
					continue;
				}

				if (shift)
				{
					objectsInside[j - 1] = objectsInside[j];
				}
			}
			insideIndex--;
		}

		// Finally, assign our biggest response to the buffer
		collisionResponse[make_uint2(launch_index.x / physicsRayStep, launch_index.y / physicsRayStep)] = largestResponse;
		
	}
}

RT_PROGRAM void perspective_camera()
{
	// Determine if we are going to use this ray for volume intersections
	bool isPhysicsRay = (launch_index.x % physicsRayStep == 0 && launch_index.y % physicsRayStep == 0);

	if (isPhysicsRay)
		ClearResponseBuffer();

	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	float3 result = make_float3( 0.0f );


	PerRayData_radiance prd;
	prd.physicsRay = isPhysicsRay;
	prd.done = false;

	prd.origin = eye;

	prd.result = make_float3(0.0, 0.0, 0.0);
	prd.importance = 1.0;
	prd.depth = 0;

	prd.numIntersections = 0;

	for (;;)
	{
		optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);
		rtTrace(top_object, ray, prd);
		result += prd.result;

		if (prd.done || !isPhysicsRay)
		{
			break;
		}

		prd.depth++;
		ray_origin = prd.origin;
	}

	if (isPhysicsRay)
		CheckIntersectionOverlap(prd, eye, ray_direction, result);

	output_buffer[launch_index] = make_color(result);
}

// Closest hit shading for the spheres
RT_PROGRAM void closest_hit_radiance()
{
	float3 hit_point = ray.origin + closestHitDist * ray.direction;
	float3 new_origin = ray.origin + (closestHitDist + 0.001) * ray.direction;
	prd_radiance.origin = new_origin;

	IntersectionData data;
	data.rigidBodyId = intersectionData.rigidBodyId;
	data.t = prd_radiance.numIntersections == 0 ? intersectionData.t : intersectionData.t + prd_radiance.intersections[prd_radiance.numIntersections - 1].t;
	data.normal = intersectionData.normal;

	prd_radiance.intersections[prd_radiance.numIntersections] = data;
	prd_radiance.numIntersections++;

	// Only shade first object we come in contact with, so no transparency
	// for now, but possible to implement later
	if (prd_radiance.numIntersections != 1)
	{
		prd_radiance.result = make_float3(0.0, 0.0, 0.0);
		return;
	}

	float3 world_geo_normal = normalize(rtTransformNormal(
										RT_OBJECT_TO_WORLD,
										geometric_normal));

	float3 world_shade_normal = normalize(rtTransformNormal(
											RT_OBJECT_TO_WORLD,
											shading_normal));

	// Handles back face rendering
	float3 ffnormal = faceforward(world_shade_normal,
									-ray.direction,
									world_geo_normal);

	float3 color = ambientColorIntensity * ambientLightColor;

	// Phong diffuse shading
	for(int i = 0; i < lights.size(); ++i) 
	{
		Light light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = __saturatef(dot( ffnormal, L));
		
		if( nDl > 0 )
		{
			// Cast a shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = 1.0f;
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray(hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			float light_attenuation = shadow_prd.attenuation;

			if (light_attenuation > 0.0f)
			{
				float3 Lc = light.color * light_attenuation;
				color += diffuseColorIntensity * nDl * Lc;

				float3 H = normalize(L - ray.direction); // half way vector
				float nDh = dot(ffnormal, H);
				if (nDh > 0)
					color += specularColorIntensity * Lc * pow(nDh, specularPower);
			}
			else
			{
				return;
			}
		}
	}

	float3 r = schlick(-dot(ffnormal, ray.direction), fresnel);
	float importance = prd_radiance.importance * optix::luminance(reflectivity);

	// reflection ray
	if (importance > importance_cutoff && prd_radiance.depth < max_depth) 
	{
		PerRayData_radiance refl_prd;
		refl_prd.importance = importance;
		refl_prd.depth = prd_radiance.depth+1;

		refl_prd.physicsRay = false;

		refl_prd.result = make_float3(0.0, 0.0, 0.0);
		refl_prd.importance = importance;
		refl_prd.depth = prd_radiance.depth+1;

		refl_prd.numIntersections = 0;

		float3 R = reflect(ray.direction, ffnormal);
		optix::Ray refl_ray( hit_point + 0.001 * R, R, radiance_ray_type, scene_epsilon );
		rtTrace(top_object, refl_ray, refl_prd);
		color += r * refl_prd.result;
	}

	prd_radiance.result = color;
}

// Miss program, stored in ray data and will be used if no intersections
// along our ray were recorded
RT_PROGRAM void miss()
{
	// No more things to raytrace!
	prd_radiance.done = true;

	float3 point = normalize(ray.direction);
	float u = atan2(point.x, point.z) / (2.0 * M_PIf) + 0.5;
	float v = point.y * 0.5 + 0.5;

	if (prd_radiance.numIntersections == 0)
	{
		prd_radiance.result = make_float3(tex2D(envmap, u, v));
	}
	else
	{
		prd_radiance.result = make_float3(0.0, 0.0, 0.0);
	}
}

RT_PROGRAM void any_hit_shadow()
{
    // Opaque shadow caster
    prd_shadow.attenuation = 0.0f;

    rtTerminateRay();
}

// Exception program, default to some known exception color
RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Exception code 0x%X at (%d, %d)\n", code, launch_index.x, launch_index.y);
	output_buffer[launch_index] = make_color(bad_color);
}

