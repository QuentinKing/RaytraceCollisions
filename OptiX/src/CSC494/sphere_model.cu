#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>
#include "RayStructs.h"

using namespace optix;

// Volumetric variables (All geometry need this)
rtDeclareVariable(IntersectionData, intersectionData, attribute intersectionData, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// Rigidbody specific variables
rtDeclareVariable(float, id, , );

// Sphere specific variables
rtDeclareVariable(float, radius, , );

// Shading variables (Technically not required, but usually used on all materials)
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


template<bool use_robust_method>
static __device__
void intersect_sphere(void)
{
	float3 O = ray.origin;
	float3 D = ray.direction;

	float b = dot(O, D);
	float c = dot(O, O) - radius * radius;
	float disc = b * b - c;
	if (disc > 0.0f) {
		float sdisc = sqrtf(disc);
		float root1 = (-b - sdisc);

		bool do_refine = false;

		float root11 = 0.0f;

		if (use_robust_method && fabsf(root1) > 10.f * radius) 
		{
			do_refine = true;
		}

		if (do_refine) 
		{
			// refine root1
			float3 O1 = O + root1 * ray.direction;
			b = dot(O1, D);
			c = dot(O1, O1) - radius * radius;
			disc = b * b - c;

			if (disc > 0.0f) {
				sdisc = sqrtf(disc);
				root11 = (-b - sdisc);
			}
		}

		float t1 = root1 + root11;
		float t2 = (-b + sdisc) + (do_refine ? root1 : 0);

		if (rtPotentialIntersection(t1))
		{
			IntersectionData entryData;
			entryData.rigidBodyId = id;
			entryData.t = t1;
			entryData.normal = (O + (root1 + root11)*D) / radius;
			intersectionData = entryData;

			shading_normal = geometric_normal = (O + (root1 + root11)*D) / radius;
			rtReportIntersection(0);
		}

		if (rtPotentialIntersection(t2))
		{
			IntersectionData exitData;
			exitData.rigidBodyId = id;
			exitData.t = t2;
			exitData.normal = (O + t2*D)/radius;
			intersectionData = exitData;

			shading_normal = geometric_normal = (O + t2*D)/radius;
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void intersect(int primIdx)
{
	intersect_sphere<false>();
}


RT_PROGRAM void robust_intersect(int primIdx)
{
	intersect_sphere<true>();
}


RT_PROGRAM void bounds(int, float result[6])
{
	const float3 rad = make_float3(radius);

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (rad.x > 0.0f && !isinf(rad.x)) {
		aabb->m_min = rad;
		aabb->m_max = rad;
	}
	else {
		aabb->invalidate();
	}
}

