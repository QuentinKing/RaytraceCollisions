#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "tutorial.h"
#include "Lights.h"

// Ray data
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float, closestHitDist, rtIntersectionDistance, );

// Camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float2, orthoCameraSize, , );

// Output buffers
rtBuffer<uchar4, 2>              output_buffer;
rtBuffer<uchar4, 2>              volume_buffer;

// Volumetric variables
rtDeclareVariable(bool, ignore_intersection, attribute ignore_intersection, );
rtDeclareVariable(float2, t_values, attribute t_values, );

// Shading values
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );

// Light values
rtDeclareVariable(float3, ambientLightColor, , );
rtBuffer<Light> lights; 

// Material values
rtDeclareVariable(float3, ambientColorIntensity, , );
rtDeclareVariable(float3, diffuseColorIntensity, , );
rtDeclareVariable(float3, specularColorIntensity, , );
rtDeclareVariable(float,   specularPower, , );


// Scene values
rtDeclareVariable(float3, bg_color, , );


//
// Collision volume functions
//

// Checks given a list of entry and exit points if any of them overlap
// indicating that an intersection has occured for the current ray
bool CheckIntersectionOverlap(PerRayData_radiance prd)
{
	float2 screen = make_float2(output_buffer.size());
	float2 pixelSize = orthoCameraSize / screen;
	for (int i = 0; i < prd.numIntersections; i++)
	{
		float2 firstInterval = prd.intersections[i];
		for (int j = i + 1; j < prd.numIntersections; j++)
		{
			float2 secondInterval = prd.intersections[j];
			if (firstInterval.x <= secondInterval.y && secondInterval.x <= firstInterval.y)
			{
				// Compute intersection volume and save it to our buffer
				float volume = (firstInterval.y - secondInterval.x) * pixelSize.x * pixelSize.y;
				float col = volume * screen.x * screen.y * 0.05f; // Compute a relevant color value for the buffer
				volume_buffer[launch_index] = make_color(make_float3(col+0.1, 0, 0));
				return true;
			}
		}
	}
	volume_buffer[launch_index] = prd.numIntersections > 0 ? make_color(make_float3(0.1, 0.1, 0.1)) : make_color(make_float3(0, 0, 0));
	return false;
}


//
// Camera functions
//

// Perspective camera
RT_PROGRAM void perspective_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;
	prd.numIntersections = 0;
	prd.closestTval = 999999.0f;

	rtTrace(top_object, ray, prd);

	volume_buffer[launch_index] = make_color(make_float3(0, 0, 0));
	if (prd.numIntersections > 0)
	{
		// Check for intersections (and fill in the intersection buffer)
		CheckIntersectionOverlap(prd);

		// Shade the object with the properties we saved while raycasting
		output_buffer[launch_index] = make_color(prd.result);
	}
	else
	{
		output_buffer[launch_index] = make_color(prd.missColor);
	}
}

// Orthographic camera (easier calculations for intersection volumes)
RT_PROGRAM void orthographic_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye + d.x*U*orthoCameraSize.x + d.y*V*orthoCameraSize.y;
	float3 ray_direction = normalize(W);

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;
	prd.numIntersections = 0;
	prd.closestTval = 999999.0f;

	prd.closestShadingNormal = make_float3(0.0, 1.0, 0.0);

	rtTrace(top_object, ray, prd);

	volume_buffer[launch_index] = make_color(make_float3(0, 0, 0));
	if (prd.numIntersections > 0)
	{
		// Check for intersections (and fill in the intersection buffer)
		CheckIntersectionOverlap(prd);

		// Shade the object with the properties we saved while raycasting
		// This is essentially the closest hit shader
		output_buffer[launch_index] = make_color(prd.result);
	}
	else
	{
		output_buffer[launch_index] = make_color(prd.missColor);
	}
}


//
// Raytrace functions
//

// Miss program, stored in ray data and will be used if no intersections
// along our ray were recorded
RT_PROGRAM void miss()
{
	prd_radiance.missColor = bg_color;
}


// Any hit program, store depth value and potential shading properties
RT_PROGRAM void any_hit()
{
	// Record our intersection values
	if (prd_radiance.numIntersections < INTERSECTION_SAMPLES)
	{
		prd_radiance.intersections[prd_radiance.numIntersections] = t_values;
		prd_radiance.numIntersections++;

		// Is this the closest object we have seen so far?
		if (min(t_values.x, t_values.y) < prd_radiance.closestTval)
		{
			// Update shading properties since this is now the closest object
			prd_radiance.closestTval = min(t_values.x, t_values.y);
			prd_radiance.closestShadingNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
		}
	}

	if (ignore_intersection)
		rtIgnoreIntersection();
}

// Closest hit shading for the spheres
RT_PROGRAM void closest_hit_radiance_sphere()
{
	float3 world_geo_normal = normalize(rtTransformNormal(
										RT_OBJECT_TO_WORLD,
										geometric_normal));

	float3 world_shade_normal = normalize(rtTransformNormal(
											RT_OBJECT_TO_WORLD,
											shading_normal));

	// Handles back face rendering
	float3 ffnormal = faceforward(world_shade_normal,
									-ray.direction,
									world_geo_normal);

	float3 color = ambientColorIntensity * ambientLightColor;
 
	float3 hit_point = ray.origin + closestHitDist * ray.direction;

	// Phong diffuse shading
	for(int i = 0; i < lights.size(); ++i) 
	{
		Light light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = __saturatef(dot( ffnormal, L));
		
		if( nDl > 0 )
		{
			// Cast a shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = 1.0f;
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray(hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			float light_attenuation = shadow_prd.attenuation;

			if (light_attenuation > 0.0f)
			{
				float3 Lc = light.color * light_attenuation;
				color += diffuseColorIntensity * nDl * Lc;

				float3 H = normalize(L - ray.direction); // half way vector
				float nDh = dot(ffnormal, H);
				if (nDh > 0)
					color += specularColorIntensity * Lc * pow(nDh, specularPower);
			}
		}
	}

	prd_radiance.result = color;
}

RT_PROGRAM void any_hit_shadow()
{
    // Opaque shadow caster
    prd_shadow.attenuation = 0.0f;

    rtTerminateRay();
}

// Closest hit shading for the plane 
RT_PROGRAM void closest_hit_radiance_plane()
{
  float3 world_geo_normal = normalize(rtTransformNormal(
										RT_OBJECT_TO_WORLD,
										geometric_normal));

	float3 world_shade_normal = normalize(rtTransformNormal(
											RT_OBJECT_TO_WORLD,
											shading_normal));

	// Handles back face rendering
	float3 ffnormal = faceforward(world_shade_normal,
									-ray.direction,
									world_geo_normal);

	float3 color = ambientColorIntensity * ambientLightColor;
 
	float3 hit_point = ray.origin + closestHitDist * ray.direction;

	// Phong diffuse shading
	for(int i = 0; i < lights.size(); ++i) 
	{
		Light light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = __saturatef(dot( ffnormal, L));
		
		if( nDl > 0 )
		{
			// Cast a shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = 1.0f;
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray(hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			float light_attenuation = shadow_prd.attenuation;

			if (light_attenuation > 0.0f)
			{
				float3 Lc = light.color * light_attenuation;
				color += diffuseColorIntensity * nDl * Lc;

				float3 H = normalize(L - ray.direction); // half way vector
				float nDh = dot(ffnormal, H);
				if (nDh > 0)
					color += specularColorIntensity * Lc * pow(nDh, specularPower);
			}
		}
	}

	prd_radiance.result = color;
}

// Closest hit shading for the spheres
RT_PROGRAM void closest_hit_radiance_box()
{
	float3 world_geo_normal = normalize(rtTransformNormal(
										RT_OBJECT_TO_WORLD,
										geometric_normal));

	float3 world_shade_normal = normalize(rtTransformNormal(
											RT_OBJECT_TO_WORLD,
											shading_normal));

	// Handles back face rendering
	float3 ffnormal = faceforward(world_shade_normal,
									-ray.direction,
									world_geo_normal);

	float3 color = ambientColorIntensity * ambientLightColor;
 
	float3 hit_point = ray.origin + closestHitDist * ray.direction;

	// Phong diffuse shading
	for(int i = 0; i < lights.size(); ++i) 
	{
		Light light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = __saturatef(dot( ffnormal, L));
		
		if( nDl > 0 )
		{
			// Cast a shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = 1.0f;
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray(hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			float light_attenuation = shadow_prd.attenuation;

			if (light_attenuation > 0.0f)
			{
				float3 Lc = light.color * light_attenuation;
				color += diffuseColorIntensity * nDl * Lc;

				float3 H = normalize(L - ray.direction); // half way vector
				float nDh = dot(ffnormal, H);
				if (nDh > 0)
					color += specularColorIntensity * Lc * pow(nDh, specularPower);
			}
		}
	}

	prd_radiance.result = color;
}

// Exception program, deafult to some known exception color
RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}
