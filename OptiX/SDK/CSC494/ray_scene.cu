#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "tutorial.h"

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );


//
// Pinhole camera implementation
//
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float2, orthoCameraSize, , );
rtBuffer<uchar4, 2>              output_buffer;
rtBuffer<uchar4, 2>              volume_buffer;

// Volumetric variables
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float2, t_values, attribute t_values, );

// Checks given a list of entry and exit points if any of them overlap
// indicating that an intersection has occured for the current ray
bool CheckIntersectionOverlap(PerRayData_radiance prd)
{
	float2 screen = make_float2(output_buffer.size());
	float2 pixelSize = orthoCameraSize / screen;
	for (int i = 0; i < prd.numIntersections; i++)
	{
		float2 firstInterval = prd.intersections[i];
		for (int j = i + 1; j < prd.numIntersections; j++)
		{
			float2 secondInterval = prd.intersections[j];
			if (firstInterval.x <= secondInterval.y && secondInterval.x <= firstInterval.y)
			{
				// Compute intersection volume and save it to our buffer
				float volume = (firstInterval.y - secondInterval.x) * pixelSize.x * pixelSize.y;
				float col = volume * screen.x * screen.y * 0.05f; // Compute a relevant color value for the buffer
				volume_buffer[launch_index] = make_color(make_float3(col+0.1, 0, 0));
				return true;
			}
		}
	}
	volume_buffer[launch_index] = prd.numIntersections > 0 ? make_color(make_float3(0.1, 0.1, 0.1)) : make_color(make_float3(0, 0, 0));
	return false;
}

// Perspective camera (Not in use currently)
RT_PROGRAM void perspective_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;
	prd.numIntersections = 0;
	prd.closestTval = 999999.0f;

	rtTrace(top_object, ray, prd);

	volume_buffer[launch_index] = make_color(make_float3(0, 0, 0));
	if (prd.numIntersections > 0)
	{
		// Check for intersections (and fill in the intersection buffer)
		CheckIntersectionOverlap(prd);

		// Shade the object with the properties we saved while raycasting
		output_buffer[launch_index] = make_color(prd.closestShadingNormal);
	}
	else
	{
		output_buffer[launch_index] = make_color(prd.result);
	}
}

// Orthographic camera (easier calculations for intersection volumes)
RT_PROGRAM void orthographic_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye + d.x*U*orthoCameraSize.x + d.y*V*orthoCameraSize.y;
	float3 ray_direction = normalize(W);

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;
	prd.numIntersections = 0;
	prd.closestTval = 999999.0f;

	prd.closestShadingNormal = make_float3(0.0, 1.0, 0.0);

	rtTrace(top_object, ray, prd);

	volume_buffer[launch_index] = make_color(make_float3(0, 0, 0));
	if (prd.numIntersections > 0)
	{
		// Check for intersections (and fill in the intersection buffer)
		CheckIntersectionOverlap(prd);

		// Shade the object with the properties we saved while raycasting
		output_buffer[launch_index] = make_color(prd.closestShadingNormal);
	}
	else
	{
		output_buffer[launch_index] = make_color(prd.result);
	}
}

//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, bg_color, , );
RT_PROGRAM void miss()
{
	prd_radiance.result = bg_color;
}


//
// Returns shading normal as the surface shading result
// 
RT_PROGRAM void closest_hit_radiance0()
{
	prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;
}

//
// Returns shading normal as the surface shading result
// 
RT_PROGRAM void closest_hit_radiance1()
{
	prd_radiance.result = make_float3(1.0f, 1.0f, 1.0f);
}

// Any hit program
RT_PROGRAM void any_hit()
{
	// Record our intersection values
	if (prd_radiance.numIntersections < INTERSECTION_SAMPLES)
	{
		prd_radiance.intersections[prd_radiance.numIntersections] = t_values;
		prd_radiance.numIntersections++;

		// TODO: Don't like the float2 type here
		if (t_values.x < prd_radiance.closestTval)
		{
			// Update shading properties since this is now the closest object
			prd_radiance.closestTval = t_values.x;
			prd_radiance.closestShadingNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;
		}
	}

	rtIgnoreIntersection();
}

//
// Set pixel to solid color upon failur
//
RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}
