#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "RayStructs.h"
#include "BufferStructs.h"

// Ray data
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float, closestHitDist, rtIntersectionDistance, );
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(int, max_depth, , );

// Camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float, fov, , );

// Output buffers
rtBuffer<uchar4, 2>								 output_buffer;
rtBuffer<IntersectionResponse, 2>				 collisionResponse;

// Rigidbody variables
rtDeclareVariable(int, numRigidbodies, , );
rtDeclareVariable(int, physicsRayStep, , );
rtDeclareVariable(int, physicsBufferWidth, , );
rtDeclareVariable(int, physicsBufferHeight, , );
rtBuffer<RigidbodyMotion> rigidbodyMotions; 

// Volumetric variables
rtDeclareVariable(IntersectionData, intersectionData, attribute intersectionData, );
rtDeclareVariable(float, staticTVal, attribute staticTVal, );
rtDeclareVariable(bool, ignore_intersection, attribute ignore_intersection, );

// Shading values
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );

// Light values
rtDeclareVariable(float3, ambientLightColor, , );
rtBuffer<Light> lights; 

// Material values
rtDeclareVariable(float3, ambientColorIntensity, , );
rtDeclareVariable(float3, diffuseColorIntensity, , );
rtDeclareVariable(float3, specularColorIntensity, , );
rtDeclareVariable(float3, fresnel, , );
rtDeclareVariable(float3, reflectivity, , );
rtDeclareVariable(float,   specularPower, , );

// Scene values
rtTextureSampler<float4, 2> envmap;

void ClearResponseBuffer()
{
	IntersectionResponse data;
	data.volume = 0.0f;
	data.entryId = 0;
	data.entryNormal = make_float3(0.0f,0.0f,0.0f);
	data.exitId = 0;
	data.exitNormal = make_float3(0.0f,0.0f,0.0f);
	data.entryPoint = make_float3(0.0f,0.0f,0.0f);
	data.exitPoint = make_float3(0.0f,0.0f,0.0f);
	collisionResponse[make_uint2(launch_index.x / physicsRayStep, launch_index.y / physicsRayStep)] = data;
}

// Checks given a list of entry and exit points if any of them overlap
// indicating that an intersection has occured for the current ray
void CheckIntersectionOverlap(PerRayData_radiance prd, float3 ray_origin, float3 ray_direction)
{
	float2 screen = make_float2(output_buffer.size());
	float total = 0.0f;

	for (int i = 0; i < prd.numIntersections; i++)
	{
		float2 firstInterval = make_float2(prd.intersections[i].entryTval, prd.intersections[i].exitTval);
		for (int j = i + 1; j < prd.numIntersections; j++)
		{
			float2 secondInterval = make_float2(prd.intersections[j].entryTval, prd.intersections[j].exitTval);

			// Compute intersection volume and save it to our buffer
			float intersection = max(0.0f, min(firstInterval.y, secondInterval.y) - max(firstInterval.x, secondInterval.x));
			int entryIndex = max(firstInterval.x, secondInterval.x) == firstInterval.x ? i : j;
			int exitIndex = min(firstInterval.y, secondInterval.y) == firstInterval.y ? i : j;

			float fovDelta = 1.0 / screen.x;
			float theta = fov * fovDelta;
			float phi = 90.0 - theta;
			float a = sin(theta) * prd.intersections[entryIndex].entryTval / sin(phi);
			float b = sin(theta) * prd.intersections[exitIndex].exitTval / sin(phi);
			float h = intersection;
			float volume = 0.33 * (a*a + a * b + b * b) * h;

			IntersectionResponse data;
			data.volume = volume;
			data.entryId = prd.intersections[entryIndex].rigidBodyId;
			data.entryNormal = prd.intersections[entryIndex].entryNormal;
			data.exitId = prd.intersections[exitIndex].rigidBodyId;
			data.exitNormal = prd.intersections[exitIndex].exitNormal;
			data.entryPoint = ray_origin + prd.intersections[entryIndex].entryTval * ray_direction;
			data.exitPoint = ray_origin + prd.intersections[exitIndex].exitTval * ray_direction;
			collisionResponse[make_uint2(launch_index.x / physicsRayStep, launch_index.y / physicsRayStep)] = data;

			total += volume;
		}
	}
}

RT_PROGRAM void perspective_camera()
{
	// Determine if we are going to use this ray for volume intersections
	bool isPhysicsRay = (launch_index.x % physicsRayStep == 0 && launch_index.y % physicsRayStep == 0);

	if (isPhysicsRay)
		ClearResponseBuffer();

	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;
	prd.numIntersections = 0;
	prd.hitObject = false;
	prd.closestTval = 999999.0f;

	rtTrace(top_object, ray, prd);

	if (prd.hitObject)
	{
		// Check for intersections (and fill in the intersection buffer)
		if (isPhysicsRay)
			CheckIntersectionOverlap(prd, ray_origin, ray_direction);
	}

	output_buffer[launch_index] = make_color(prd.result);
}

// Closest hit shading for the spheres
RT_PROGRAM void closest_hit_radiance()
{
	float3 world_geo_normal = normalize(rtTransformNormal(
										RT_OBJECT_TO_WORLD,
										geometric_normal));

	float3 world_shade_normal = normalize(rtTransformNormal(
											RT_OBJECT_TO_WORLD,
											shading_normal));

	// Handles back face rendering
	float3 ffnormal = faceforward(world_shade_normal,
									-ray.direction,
									world_geo_normal);

	float3 color = ambientColorIntensity * ambientLightColor;
 
	float3 hit_point = ray.origin + closestHitDist * ray.direction;

	// Phong diffuse shading
	for(int i = 0; i < lights.size(); ++i) 
	{
		Light light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = __saturatef(dot( ffnormal, L));
		
		if( nDl > 0 )
		{
			// Cast a shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = 1.0f;
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray(hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			float light_attenuation = shadow_prd.attenuation;

			if (light_attenuation > 0.0f)
			{
				float3 Lc = light.color * light_attenuation;
				color += diffuseColorIntensity * nDl * Lc;

				float3 H = normalize(L - ray.direction); // half way vector
				float nDh = dot(ffnormal, H);
				if (nDh > 0)
					color += specularColorIntensity * Lc * pow(nDh, specularPower);
			}
		}
	}

	float3 r = schlick(-dot(ffnormal, ray.direction), fresnel);
	float importance = prd_radiance.importance * optix::luminance(reflectivity);

	// reflection ray
	if (importance > importance_cutoff && prd_radiance.depth < max_depth) 
	{
		PerRayData_radiance refl_prd;
		refl_prd.importance = importance;
		refl_prd.depth = prd_radiance.depth+1;
		float3 R = reflect(ray.direction, ffnormal);
		optix::Ray refl_ray( hit_point, R, radiance_ray_type, scene_epsilon );
		rtTrace(top_object, refl_ray, refl_prd);
		color += r * refl_prd.result;
	}

	prd_radiance.result = color;
}

// Any hit program, store depth value and potential shading properties
RT_PROGRAM void any_hit()
{
	prd_radiance.hitObject = true;

	// Record our intersection values
	if (prd_radiance.numIntersections < INTERSECTION_SAMPLES)
	{
		prd_radiance.intersections[prd_radiance.numIntersections] = intersectionData;
		prd_radiance.numIntersections++;

		// Is this the closest object we have seen so far?
		if (intersectionData.entryTval < prd_radiance.closestTval)
		{
			// Update shading properties since this is now the closest object
			prd_radiance.closestTval = intersectionData.entryTval;
		}
	}

	if (ignore_intersection)
		rtIgnoreIntersection();
}

RT_PROGRAM void any_hit_static()
{
	// Record our intersection values
	prd_radiance.hitObject = true;
	prd_radiance.closestTval = min(prd_radiance.closestTval, staticTVal);
}

// Miss program, stored in ray data and will be used if no intersections
// along our ray were recorded
RT_PROGRAM void miss()
{
	float3 point = normalize(ray.direction);
	float u = atan2(point.x, point.z) / (2.0 * M_PIf) + 0.5;
	float v = point.y * 0.5 + 0.5;
	prd_radiance.result = make_float3(tex2D(envmap, u, v));
}

RT_PROGRAM void any_hit_shadow()
{
    // Opaque shadow caster
    prd_shadow.attenuation = 0.0f;

    rtTerminateRay();
}

// Exception program, deafult to some known exception color
RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Exception code 0x%X at (%d, %d)\n", code, launch_index.x, launch_index.y);
	output_buffer[launch_index] = make_color(bad_color);
}

