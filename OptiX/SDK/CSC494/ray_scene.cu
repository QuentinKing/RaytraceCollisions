#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "tutorial.h"

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );


//
// Pinhole camera implementation
//
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float2, orthoCameraSize, , );
rtBuffer<uchar4, 2>              output_buffer;

// Volumetric variables
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float2, t_values, attribute t_values, );

// Checks given a list of entry and exit points if any of them overlap
// indicating that an intersection has occured for the current ray
bool CheckIntersectionOverlap(PerRayData_radiance prd)
{
	for (int i = 0; i < prd.numIntersections; i++)
	{
		float2 firstInterval = prd.intersections[i];
		for (int j = i + 1; j < prd.numIntersections; j++)
		{
			float2 secondInterval = prd.intersections[j];
			if (firstInterval.x <= secondInterval.y && secondInterval.x <= firstInterval.y)
			{
				return true;
			}
		}
	}
	return false;
}

// Perspective camera (Not in use currently)
RT_PROGRAM void perspective_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;
	prd.numIntersections = 0;

	rtTrace(top_object, ray, prd);

	output_buffer[launch_index] = make_color(prd.result);
}

// Orthographic camera (easier calculations for intersection volumes)
RT_PROGRAM void orthographic_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye + d.x*U*orthoCameraSize.x + d.y*V*orthoCameraSize.y;
	float3 ray_direction = normalize(W);

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;
	prd.numIntersections = 0;

	rtTrace(top_object, ray, prd);

	if (prd.numIntersections > 0)
	{
		output_buffer[launch_index] = CheckIntersectionOverlap(prd) ? make_color(make_float3(0, 0, 0)) : make_color(make_float3(1, 1, 1));
	}
	else
	{
		output_buffer[launch_index] = make_color(prd.result);
	}
}

//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, bg_color, , );
RT_PROGRAM void miss()
{
	prd_radiance.result = bg_color;
}


//
// Returns shading normal as the surface shading result
// 
RT_PROGRAM void closest_hit_radiance0()
{
	prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;
}

//
// Returns shading normal as the surface shading result
// 
RT_PROGRAM void closest_hit_radiance1()
{
	prd_radiance.result = make_float3(1.0f, 1.0f, 1.0f);
}

// Any hit program
RT_PROGRAM void any_hit()
{
	// Record our intersection values
	if (prd_radiance.numIntersections < INTERSECTION_SAMPLES)
	{
		prd_radiance.intersections[prd_radiance.numIntersections] = t_values;
		prd_radiance.numIntersections++;
	}

	rtIgnoreIntersection();
}

//
// Set pixel to solid color upon failur
//
RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}
