#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>
#include "tutorial.h"

using namespace optix;

// Volumetric variables (All geometry need this)
rtDeclareVariable(IntersectionData, intersectionData, attribute intersectionData, );
rtDeclareVariable(bool, ignore_intersection, attribute ignore_intersection, );
rtDeclareVariable(float, current_closest, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// Rigidbody specific variables
rtDeclareVariable(float, id, , );

// Sphere specific variables
rtDeclareVariable(float, radius, , );

// Shading variables (Technically not required, but usually used on all materials)
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


template<bool use_robust_method>
static __device__
void intersect_sphere(void)
{
	float3 O = ray.origin;
	float3 D = ray.direction;

	float b = dot(O, D);
	float c = dot(O, O) - radius * radius;
	float disc = b * b - c;
	if (disc > 0.0f) {
		float sdisc = sqrtf(disc);
		float root1 = (-b - sdisc);

		bool do_refine = false;

		float root11 = 0.0f;

		if (use_robust_method && fabsf(root1) > 10.f * radius) 
		{
			do_refine = true;
		}

		if (do_refine) 
		{
			// refine root1
			float3 O1 = O + root1 * ray.direction;
			b = dot(O1, D);
			c = dot(O1, O1) - radius * radius;
			disc = b * b - c;

			if (disc > 0.0f) {
				sdisc = sqrtf(disc);
				root11 = (-b - sdisc);
			}
		}

		float t1 = root1 + root11;
		float t2 = (-b + sdisc) + (do_refine ? root1 : 0);

		// Always call the any hit function, so we have to report an intersection closer than 
		// the closest intersection. If we have to fudge the numbers a bit to make sure we call the any-hit
		// function, make sure we ignore the intersection so it doesn't store this value.
		bool ignore = t1 > current_closest; 
		float modified_t_value = ignore ? current_closest - 1.0f : t1;

		if (rtPotentialIntersection(modified_t_value))
		{
			ignore_intersection = ignore;

			IntersectionData data;
			data.rigidBodyId = id;
			data.entryTval = t1;
			data.exitTval = t2;
			data.entryNormal = (O + (root1 + root11)*D) / radius;
			data.exitNormal = (O + t2*D)/radius;
			intersectionData = data;

			shading_normal = geometric_normal = data.entryNormal;
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void intersect(int primIdx)
{
	intersect_sphere<false>();
}


RT_PROGRAM void robust_intersect(int primIdx)
{
	intersect_sphere<true>();
}


RT_PROGRAM void bounds(int, float result[6])
{
	const float3 rad = make_float3(radius);

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (rad.x > 0.0f && !isinf(rad.x)) {
		aabb->m_min = rad;
		aabb->m_max = rad;
	}
	else {
		aabb->invalidate();
	}
}

